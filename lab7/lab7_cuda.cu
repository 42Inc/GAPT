
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initFun(int *nf) {
    int n = threadIdx.x + blockIdx.x * blockDim.x;
    nf[n] *= 10;
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        fprintf(stderr, "USAGE: main <num_of_devices> " 
                   "<device_indices>\n");
        return -1;
    }

    int *info_devs = (int *) calloc(argc - 2, sizeof(int));

    info_devs[0] = atoi(argv[1]);
    for (int i = 1; i < argc - 2; i++) {
        info_devs[i] = atoi(argv[i + 1]);
    }

    int N = atoi(argv[4]);
    float elapsed_time;

    //printf("%d\n", N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipStream_t* streams;
    int **nfd = (int **) calloc(info_devs[0], sizeof(int *));
    int **nfh = (int **) calloc(info_devs[0], sizeof(int *));

    streams = (hipStream_t *) calloc(info_devs[0], sizeof(hipStream_t));

    hipEventRecord(start, 0);

    for (int i = 0; i < info_devs[0]; i++) {
        hipSetDevice(info_devs[i + 1]);
        hipStreamCreate(&streams[i]);

        hipMalloc((void **) &nfd[i], (N / info_devs[0]) * sizeof(int));
        hipHostMalloc((void **) &nfh[i], (N / info_devs[0]) * sizeof(int), hipHostMallocDefault);
     
        for (int n = 0; n < N / info_devs[0]; n++)
            nfh[i][n] = n + i * N / info_devs[0];

        hipMemcpyAsync(nfd[i], nfh[i], (N / info_devs[0]) * sizeof(int),
                    hipMemcpyHostToDevice, streams[i]);
       
        initFun <<< N / info_devs[0] / 32, 32, 0, streams[i] >>>(nfd[i]);

        hipMemcpyAsync(nfh[i], nfd[i], (N / info_devs[0]) * sizeof(int),
                    hipMemcpyDeviceToHost, streams[i]);
    }

    for (int i = 0; i < info_devs[0]; i++) {
        hipSetDevice(info_devs[i + 1]);
        hipStreamSynchronize(streams[i]);
        
        //for (int n = 0; n < N / info_devs[0]; n++)
            //fprintf(stderr, "nfh[%d][%d] = %d\n", i, n, nfh[i][n]);
    }

    hipSetDevice(info_devs[1]);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("%f\n", elapsed_time);

    for (int i = 0; i < info_devs[0]; i++) {   
        hipStreamDestroy(streams[i]);
        hipFree(nfd[i]);
        hipHostFree(nfh[i]);
        hipDeviceReset();
    }

    return 0;
}