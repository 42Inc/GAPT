
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 10)

int main() {
    srand(time(NULL));

    int *dev_a;
    int *dev_a_p;
    int *h_a, *h_b;
    int *h_a_p, *h_b_p;

    float elapsed_time;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    h_a = (int *) malloc(FULL_DATA_SIZE * sizeof(int));
    h_b = (int *) malloc(FULL_DATA_SIZE * sizeof(int));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        h_a[i] = rand() % 1000;
    }

    hipMalloc((void **) &dev_a, FULL_DATA_SIZE * sizeof(int));

    hipEventRecord(start, 0);
    hipMemcpy(dev_a, h_a, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time copy common memory (host->device): %f\n", elapsed_time);

    hipEventRecord(start, 0);
    hipMemcpy(h_b, dev_a, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time copy common memory (device->host): %f\n", elapsed_time);

    hipHostAlloc((void **) &h_a_p, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void **) &h_b_p, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    hipMalloc((void **) &dev_a_p, FULL_DATA_SIZE * sizeof(int));

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        h_a_p[i] = rand() % 1000;
    }

    hipEventRecord(start, 0);
    hipMemcpy(dev_a_p, h_a_p, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time copy paged-locked memory (host->device): %f\n", elapsed_time);

    hipEventRecord(start, 0);
    hipMemcpy(h_b_p, dev_a_p, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("Elapsed time copy paged-locked memory (device->host): %f\n", elapsed_time);

    hipFree(dev_a);
    hipFree(dev_a_p);
    hipHostFree(h_a_p);
    hipHostFree(h_b_p);
    free(h_a);
    free(h_b);

    return 0;
}