#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdio.h>

#define N 10
#define NELEMS (N * N)

#define CUDA_CHECK_RETURN(value)                                    \
  {                                                                 \
    hipError_t _m_cudaStat = value;                                \
    if (_m_cudaStat != hipSuccess)                                 \
    {                                                               \
      fprintf(stderr, "Error %s at line %d in file %s\n",           \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
      exit(1);                                                      \
    }                                                               \
  }

double wtime()
{
  struct timeval t;
  gettimeofday(&t, NULL);
  return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

__global__ void init1(float *a, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i < n) && (j < n))
    a[i + j * n] = (float)(threadIdx.x + blockDim.y * blockIdx.x);
}

__global__ void init2(float *a, int n)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  if ((i < n) && (j < n))
    a[j + i * n] = (float)(threadIdx.y + blockDim.x * threadIdx.y);
}

__global__ void tr(const float *a, float *b, int n)
{
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  int j = threadIdx.y + blockDim.y * blockIdx.y;
  if ((i < n) && (j < n))
    b[j * n + i] = a[i * n + j];
}

int main()
{
  size_t size = sizeof(float) * NELEMS;
  double tgpu = 0, tmem = 0;
  float elapsedTime = 0;
  hipEvent_t start, stop;
  /* Allocate vectors on host */
  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  if (h_A == NULL || h_B == NULL)
  {
    fprintf(stderr, "Allocation error.\n");
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < NELEMS; ++i)
  {
    h_A[i] = rand() / (float)RAND_MAX;
  }

  /* Allocate vectors on device */
  float *d_A = NULL, *d_B = NULL;
  tmem = -wtime();
  CUDA_CHECK_RETURN(hipMalloc((void **)&d_A, size));
  CUDA_CHECK_RETURN(hipMalloc((void **)&d_B, size));

  /* Copy the host vectors to device */
  CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice))
  tmem += wtime();

  /* Launch the kernel */
  hipEventCreate(&start);
  hipEventCreate(&stop);
  tgpu = -wtime();
  int threadsPerBlock = 1024;
  int blocksPerGrid = (NELEMS + threadsPerBlock - 1) / threadsPerBlock;
  hipEventRecord(start, 0);
  tr<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  tgpu += wtime();
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  /* Copy the device vectors to host */
  tmem -= wtime();
  CUDA_CHECK_RETURN(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));
  tmem += wtime();

  for (int i = 0; i < N; ++i)
  {
  for (int j = 0; i < N; ++i)
    if (fabs(h_A[i*N +j] - h_B[j*N + i]) > 1e-5)
    {
      fprintf(stderr,
              "Result verification failed at element %d,%d! Ex: %f, Real: %f\n",
              i, h_A[i*N +j], h_B[j*N + i]);
      exit(EXIT_FAILURE);
    }
  }

  printf("Transponse\n");
  printf("GPU version (sec.): %.6lf\n", tgpu);
  printf("Memory ops. (sec.): %.6lf\n", tmem);
  printf("Total time (sec.): %.6lf\n", tgpu + tmem);
  printf("Events Time %.6f\n", elapsedTime);

  /* Launch the kernel */
  hipEventCreate(&start);
  hipEventCreate(&stop);
  tgpu = -wtime();
  hipEventRecord(start, 0);
  init1<<<blocksPerGrid, threadsPerBlock>>>(d_A, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  tgpu += wtime();
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  /* Copy the device vectors to host */
  tmem -= wtime();
  CUDA_CHECK_RETURN(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));
  tmem += wtime();

  printf("\nInit scheme 1\n");
  printf("GPU version (sec.): %.6lf\n", tgpu);
  printf("Events Time %.6f\n", elapsedTime);

  /* Launch the kernel */
  hipEventCreate(&start);
  hipEventCreate(&stop);
  tgpu = -wtime();
  hipEventRecord(start, 0);
  init2<<<blocksPerGrid, threadsPerBlock>>>(d_B, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  tgpu += wtime();
  CUDA_CHECK_RETURN(hipGetLastError());
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  /* Copy the device vectors to host */
  tmem -= wtime();
  CUDA_CHECK_RETURN(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost));
  tmem += wtime();

  printf("\nInit scheme 2\n");
  printf("GPU version (sec.): %.6lf\n", tgpu);
  printf("Events Time %.6f\n", elapsedTime);

  hipFree(d_A);
  hipFree(d_B);
  free(h_A);
  free(h_B);
  hipDeviceReset();
  return 0;
}
